#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <math.h>
#include "devHostConstants.h"
//#include "devGlobalVars.h"
__constant__ float DEV_EXP = EXP_SUM;
__constant__ int DEV_N_NEURONS = N_NEURONS, DEV_NE = NE;

/* gpu kernel */
__global__ void kernel(int nSpks, int *dev_spkNeuronId, float *dev_conVec, 
		       float *gE_data, float *gI_data){
  //
  int nNeuron = blockIdx.x;// + blockDim.x * blockIdx.x;
  int i;
  /* compute squares of entries in data array */
  // !!!!! neurons ids start from ZERO  !!!!!! 
  if(nNeuron <= DEV_N_NEURONS) {
    //g_result[nNeuron] = g_data[nNeuron];
    gE_data[nNeuron] *= DEV_EXP;
    gI_data[nNeuron] *= DEV_EXP;
    if(nSpks > 0){
      for(i = 0; i < nSpks; ++i) { //  
	if(dev_spkNeuronId[i] <= DEV_NE) {
	  gE_data[nNeuron] += dev_conVec[dev_spkNeuronId[i] * (DEV_N_NEURONS + 1) + nNeuron]; //optimize !!!! gEI_E
	}
	else {
	  gI_data[nNeuron] += dev_conVec[dev_spkNeuronId[i] * (DEV_N_NEURONS + 1) + nNeuron]; //optimize !!!! gEI_I
	}
      }
    }
  }
}
/* only use extern if calling code is C */
extern "C" 
{
  /* driver for kernel */
  void cudakernel(int nSpks, int *dev_spkNeuronId, float *dev_conVec,
		  float *gE_data, float *gI_data){
    /* choose 256 threads per block for high occupancy */
    //int ThreadsPerBlock = 256;
      /* find number of blocks */
    //    int BlocksPerGrid = (DEV_N_NEURONS+ThreadsPerBlock-1)/ThreadsPerBlock;
      /* invoke device on this block/thread grid */
    // kernel <<< BlocksPerGrid, ThreadsPerBlock >>> (DEV_N_NEURONS, nSpks, dev_spkNeuronId, dev_conVec, 
    // 						   g_data, g_result);
    kernel <<< 8, 1>>> (nSpks, dev_spkNeuronId, dev_conVec, gE_data, gI_data);
  }
}
