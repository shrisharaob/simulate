#include "hip/hip_runtime.h"
// synaptic input optimized network equations
#include <hip/hip_runtime.h>
#include "devHostConstants.h"
//extern double *input_cur, *iSynap, *iBg, *outvars;

__device__ float alpha_n(float vm);
__device__ float alpha_m(float vm);
__device__ float alpha_h(float vm);
__device__ float beta_n(float vm);
__device__ float beta_m(float vm);
__device__ float beta_h(float vm);

__device__ float  alpha_n(float vm) {
  float out;
  if(vm != -34) { 
    out = 0.1 * (vm + 34) / (1 - exp(-0.1 * (vm + 34)));
  }
  else {
    out = 0.1;
  }
  return out;
}

__device__ float beta_n(float vm) {
  float out;
  out = 1.25 * exp(- (vm + 44) / 80);
  return out;
}

__device__ float alpha_m(float vm) {
  float out;
  if(vm != -30) { 
    out = 0.1 * (vm + 30) / (1 - exp(-0.1 * (vm + 30)));
  }
  else {
    out = 1;
  }
  return out;
}

__device__ float beta_m(float vm) {
  float out;
  out = 4 * exp(-(vm + 55) / 18);
  return out;
}

__device__ float alpha_h(float vm) {
  float out;
  out = 0.7 * exp(- (vm + 44) / 20);
  return out;
}

__device__ float beta_h(float vm) {
  float out;
  out = 10 / (exp(-0.1 * (vm + 14)) + 1);
  return out;
  }

__device__ float m_inf(float vm) {
  float out, temp;
  temp = alpha_m(vm);
  out = temp / (temp + beta_m(vm));
  return out;
}

//z is the gating varible of the adaptation current
__device__ float z_inf(float(vm)) {
  float out;
  out = 1 / (1 + exp(-0.7 *(vm + 30)));
  return out;
}

extern float dt, *iSynap;
// m_inf 
// stateVar = [vm, n, z, h]
// z - gating variable of the adaptation current
__device__ void derivs(float t, float stateVar[], float dydx[]) {
  int tIdx, kNeuron, colNo;
  double cur = 0;
  tIdx = (int)(t / dt) + 1;
  for(kNeuron = 1; kNeuron < N_Neurons + 1; ++kNeuron) {
    colNo = (kNeuron - 1) * N_StateVars;
    /* if(kNeuron == 1 && t >= 30 && t <= 35) {  */
    /*  cur = 10;//input_cur[tIdx];  */
    /*  }  */
    /* else {cur = 0;}  */
    //     cur = 10;
    cur = 0.25 * sqrt(K);
    //       cur=2.8;
    //    printf("\n ICur : %f", cur);
    if (kNeuron <= NE) { 
      dydx[1 + colNo] =  1/Cm * (cur 
                                 - G_Na * pow(m_inf(stateVar[1 + colNo]), 3) * stateVar[4 + colNo] * (stateVar[1 + colNo] - E_Na) 
                                 - G_K * pow(stateVar[2 + colNo], 4) * (stateVar[1 + colNo] - E_K) 
                                 - G_L_E * (stateVar[1 + colNo] - E_L)
                                 - G_adapt * stateVar[3 + colNo] * (stateVar[1 + colNo] - E_K) + iSynap[kNeuron]);// iBg[kNeuron]);//+ iFF[kNeuron]); // N = [NE; NI]
      }
      else {
        dydx[1 + colNo] =  1/Cm * (cur  
                                   - G_Na * pow(m_inf(stateVar[1 + colNo]), 3) * stateVar[4 + colNo] * (stateVar[1 + colNo] - E_Na) 
                                   - G_K * pow(stateVar[2 + colNo], 4) * (stateVar[1 + colNo] - E_K) 
                                   - G_L_I * (stateVar[1 + colNo] - E_L)
                                   - G_adapt * stateVar[3 + colNo] * (stateVar[1 + colNo] - E_K) + iSynap[kNeuron]); // + iBg[kNeuron]);//+ iFF[kNeuron]); // N = [NE; NI]
      }
     
    dydx[2 + colNo] = alpha_n(stateVar[1 + colNo]) * (1 - stateVar[2 + colNo]) 
                      - beta_n(stateVar[1 + colNo]) * stateVar[2 + colNo];
  
    dydx[3 + colNo] = 1 / Tau_adapt * (z_inf(stateVar[1 + colNo]) - stateVar[3 + colNo]);
    
    dydx[4 + colNo] = alpha_h(stateVar[1 + colNo]) * (1 - stateVar[4 + colNo]) 
                      - beta_h(stateVar[1 + colNo]) * stateVar[4 + colNo];
  }

}
